#ifndef __APPLE__
#include <cmath>
#include <algorithm>
#endif

#ifdef __APPLE__
// On macOS, avoid CUDA entirely and use standard C++ math
#include <cmath>
#include <algorithm>
#define __global__
#define __host__
#define __device__
#define blockIdx (dim3{0,0,0})
#define blockDim (dim3{1,1,1})
#define threadIdx (dim3{0,0,0})
struct dim3 { int x, y, z; };
#else
// Real CUDA for Windows/Linux
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#endif

__global__ void ContrastSatVolumeKernel(int p_Width, int p_Height,
                                        float* p_RgbGammas, float* p_CymGammas,
                                        float* p_RgbMidgreys, float* p_CymMidgreys,
                                        float* p_Tilts,
                                        const float* p_Input, float* p_Output)
{
#ifndef __APPLE__
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < p_Width) && (y < p_Height))
    {
        const int index = ((y * p_Width) + x) * 4;

        // Extract RGBA values
        float r = p_Input[index + 0];
        float g = p_Input[index + 1];
        float b = p_Input[index + 2];
        float a = p_Input[index + 3];
        
        // Extract parameters from arrays
        float gammaR = p_RgbGammas[0], gammaG = p_RgbGammas[1], gammaB = p_RgbGammas[2];
        float gammaC = p_CymGammas[0], gammaM = p_CymGammas[1], gammaY = p_CymGammas[2];
        float midgreyR = p_RgbMidgreys[0], midgreyG = p_RgbMidgreys[1], midgreyB = p_RgbMidgreys[2];
        float midgreyC = p_CymMidgreys[0], midgreyM = p_CymMidgreys[1], midgreyY = p_CymMidgreys[2];
        float tiltCR = p_Tilts[0], tiltGM = p_Tilts[1], tiltBY = p_Tilts[2];
        
        // Apply contrast to ENTIRE RGB using different gammas (like CPU applyGammaContrastFloat3)
        
        // RGB adjustments - apply each gamma to all RGB channels
        float safeR = fmaxf(0.001f, fminf(0.999f, r));
        float safeG = fmaxf(0.001f, fminf(0.999f, g));
        float safeB = fmaxf(0.001f, fminf(0.999f, b));
        
        // Red gamma applied to all RGB channels
        float safeMidgreyR = fmaxf(0.001f, fminf(0.999f, midgreyR));
        float recipMidgreyR = 1.0f / safeMidgreyR;
        float rAdj_R = powf(safeR * recipMidgreyR, gammaR) * safeMidgreyR;
        float rAdj_G = powf(safeG * recipMidgreyR, gammaR) * safeMidgreyR;
        float rAdj_B = powf(safeB * recipMidgreyR, gammaR) * safeMidgreyR;
        
        // Green gamma applied to all RGB channels
        float safeMidgreyG = fmaxf(0.001f, fminf(0.999f, midgreyG));
        float recipMidgreyG = 1.0f / safeMidgreyG;
        float gAdj_R = powf(safeR * recipMidgreyG, gammaG) * safeMidgreyG;
        float gAdj_G = powf(safeG * recipMidgreyG, gammaG) * safeMidgreyG;
        float gAdj_B = powf(safeB * recipMidgreyG, gammaG) * safeMidgreyG;
        
        // Blue gamma applied to all RGB channels
        float safeMidgreyB = fmaxf(0.001f, fminf(0.999f, midgreyB));
        float recipMidgreyB = 1.0f / safeMidgreyB;
        float bAdj_R = powf(safeR * recipMidgreyB, gammaB) * safeMidgreyB;
        float bAdj_G = powf(safeG * recipMidgreyB, gammaB) * safeMidgreyB;
        float bAdj_B = powf(safeB * recipMidgreyB, gammaB) * safeMidgreyB;
        
        // CYM adjustments - apply each gamma to all RGB channels
        float safeMidgreyC = fmaxf(0.001f, fminf(0.999f, midgreyC));
        float recipMidgreyC = 1.0f / safeMidgreyC;
        float cAdj_R = powf(safeR * recipMidgreyC, gammaC) * safeMidgreyC;
        float cAdj_G = powf(safeG * recipMidgreyC, gammaC) * safeMidgreyC;
        float cAdj_B = powf(safeB * recipMidgreyC, gammaC) * safeMidgreyC;
        
        float safeMidgreyM = fmaxf(0.001f, fminf(0.999f, midgreyM));
        float recipMidgreyM = 1.0f / safeMidgreyM;
        float mAdj_R = powf(safeR * recipMidgreyM, gammaM) * safeMidgreyM;
        float mAdj_G = powf(safeG * recipMidgreyM, gammaM) * safeMidgreyM;
        float mAdj_B = powf(safeB * recipMidgreyM, gammaM) * safeMidgreyM;
        
        float safeMidgreyY = fmaxf(0.001f, fminf(0.999f, midgreyY));
        float recipMidgreyY = 1.0f / safeMidgreyY;
        float yAdj_R = powf(safeR * recipMidgreyY, gammaY) * safeMidgreyY;
        float yAdj_G = powf(safeG * recipMidgreyY, gammaY) * safeMidgreyY;
        float yAdj_B = powf(safeB * recipMidgreyY, gammaY) * safeMidgreyY;
        
        // Mix results - each output channel gets specific adjustment blend
        float finalR = cAdj_R + (rAdj_R - cAdj_R) * tiltCR;  // Red output uses Red gamma vs Cyan gamma
        float finalG = mAdj_G + (gAdj_G - mAdj_G) * tiltGM;  // Green output uses Green gamma vs Magenta gamma  
        float finalB = yAdj_B + (bAdj_B - yAdj_B) * tiltBY;  // Blue output uses Blue gamma vs Yellow gamma

        // Clamp and output
        p_Output[index + 0] = fmaxf(0.0f, fminf(1.0f, finalR));
        p_Output[index + 1] = fmaxf(0.0f, fminf(1.0f, finalG));
        p_Output[index + 2] = fmaxf(0.0f, fminf(1.0f, finalB));
        p_Output[index + 3] = a;
    }
#endif
}

__global__ void GainAdjustKernel(int p_Width, int p_Height, float p_GainR, float p_GainG, float p_GainB, float p_GainA, const float* p_Input, float* p_Output)
{
#ifndef __APPLE__
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < p_Width) && (y < p_Height))
    {
        const int index = ((y * p_Width) + x) * 4;

        p_Output[index + 0] = p_Input[index + 0] * p_GainR;
        p_Output[index + 1] = p_Input[index + 1] * p_GainG;
        p_Output[index + 2] = p_Input[index + 2] * p_GainB;
        p_Output[index + 3] = p_Input[index + 3] * p_GainA;
    }
#endif
}

#ifdef __APPLE__
// Stub implementations for macOS (does nothing)
void RunComplexCudaKernel(void* p_Stream, int p_Width, int p_Height,
                         float* p_RgbGammas, float* p_CymGammas,
                         float* p_RgbMidgreys, float* p_CymMidgreys,
                         float* p_Tilts,
                         const float* p_Input, float* p_Output)
{
    // Do nothing on macOS - Metal will be used instead
}

void RunCudaKernel(void* p_Stream, int p_Width, int p_Height, float* p_Gain, const float* p_Input, float* p_Output)
{
    // Do nothing on macOS - Metal will be used instead
}

#else
// Real CUDA implementations for Windows/Linux

void RunComplexCudaKernel(void* p_Stream, int p_Width, int p_Height,
                         float* p_RgbGammas, float* p_CymGammas,
                         float* p_RgbMidgreys, float* p_CymMidgreys,
                         float* p_Tilts,
                         const float* p_Input, float* p_Output)
{
    // SAFETY CHECK: Verify inputs are valid
    if (!p_Input || !p_Output || p_Width <= 0 || p_Height <= 0) {
        return;
    }
    
    if (!p_RgbGammas || !p_CymGammas || !p_RgbMidgreys || !p_CymMidgreys || !p_Tilts) {
        return;
    }

    dim3 threads(16, 16, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), ((p_Height + threads.y - 1) / threads.y), 1);
    hipStream_t stream = static_cast<hipStream_t>(p_Stream);

    // Use simpler approach - pass arrays directly to kernel
    ContrastSatVolumeKernel<<<blocks, threads, 0, stream>>>(p_Width, p_Height,
                                                            p_RgbGammas, p_CymGammas,
                                                            p_RgbMidgreys, p_CymMidgreys,
                                                            p_Tilts,
                                                            p_Input, p_Output);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }
    
    // Synchronize to ensure completion
    if (stream) {
        hipStreamSynchronize(stream);
    } else {
        hipDeviceSynchronize();
    }
}

void RunCudaKernel(void* p_Stream, int p_Width, int p_Height, 
                   const float* p_Input, float* p_Output)
{
#ifdef __APPLE__
    // On macOS, just do CPU passthrough
    for (int i = 0; i < p_Width * p_Height * 4; i++) {
        p_Output[i] = p_Input[i];
    }
#else
    // Convert to old complex parameters for now - TODO: implement OpenDRT
    float rgbGammas[3] = {1.0f, 1.0f, 1.0f}; // No adjustment for now
    float cymGammas[3] = {1.0f, 1.0f, 1.0f};
    float rgbMidgreys[3] = {0.18f, 0.18f, 0.18f};
    float cymMidgreys[3] = {0.18f, 0.18f, 0.18f};
    float tilts[3] = {1.0f, 1.0f, 1.0f}; // Pure RGB mode
    
    RunComplexCudaKernel(p_Stream, p_Width, p_Height, 
                        rgbGammas, cymGammas, rgbMidgreys, cymMidgreys, tilts,
                        p_Input, p_Output);
#endif
}

#endif
