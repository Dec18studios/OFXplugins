// Mac implementation using CPU fallback (no CUDA support on Mac)

#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>

void RunComplexCudaKernel(void* p_Stream, int p_Width, int p_Height,
                         float* p_RgbGammas, float* p_CymGammas,
                         float* p_RgbMidgreys, float* p_CymMidgreys,
                         float* p_Tilts,
                         const float* p_Input, float* p_Output)
{
    // Mac fallback - run on CPU since CUDA is not available
    // This provides compatibility but won't be GPU accelerated
    
    for (int y = 0; y < p_Height; ++y) {
        for (int x = 0; x < p_Width; ++x) {
            const int index = ((y * p_Width) + x) * 4;

            // Extract RGBA values
            float r = p_Input[index + 0];
            float g = p_Input[index + 1];
            float b = p_Input[index + 2];
            float a = p_Input[index + 3];
            
            // Extract parameters from arrays
            float gammaR = p_RgbGammas[0], gammaG = p_RgbGammas[1], gammaB = p_RgbGammas[2];
            float gammaC = p_CymGammas[0], gammaM = p_CymGammas[1], gammaY = p_CymGammas[2];
            float midgreyR = p_RgbMidgreys[0], midgreyG = p_RgbMidgreys[1], midgreyB = p_RgbMidgreys[2];
            float midgreyC = p_CymMidgreys[0], midgreyM = p_CymMidgreys[1], midgreyY = p_CymMidgreys[2];
            float tiltCR = p_Tilts[0], tiltGM = p_Tilts[1], tiltBY = p_Tilts[2];
            
            // RGB adjustments - apply each gamma to all RGB channels
            float safeR = std::max(0.001f, std::min(0.999f, r));
            float safeG = std::max(0.001f, std::min(0.999f, g));
            float safeB = std::max(0.001f, std::min(0.999f, b));
            
            // Red gamma applied to all RGB channels
            float safeMidgreyR = std::max(0.001f, std::min(0.999f, midgreyR));
            float recipMidgreyR = 1.0f / safeMidgreyR;
            float rAdj_R = std::pow(safeR * recipMidgreyR, gammaR) * safeMidgreyR;
            float rAdj_G = std::pow(safeG * recipMidgreyR, gammaR) * safeMidgreyR;
            float rAdj_B = std::pow(safeB * recipMidgreyR, gammaR) * safeMidgreyR;
            
            // Green gamma applied to all RGB channels
            float safeMidgreyG = std::max(0.001f, std::min(0.999f, midgreyG));
            float recipMidgreyG = 1.0f / safeMidgreyG;
            float gAdj_R = std::pow(safeR * recipMidgreyG, gammaG) * safeMidgreyG;
            float gAdj_G = std::pow(safeG * recipMidgreyG, gammaG) * safeMidgreyG;
            float gAdj_B = std::pow(safeB * recipMidgreyG, gammaG) * safeMidgreyG;
            
            // Blue gamma applied to all RGB channels
            float safeMidgreyB = std::max(0.001f, std::min(0.999f, midgreyB));
            float recipMidgreyB = 1.0f / safeMidgreyB;
            float bAdj_R = std::pow(safeR * recipMidgreyB, gammaB) * safeMidgreyB;
            float bAdj_G = std::pow(safeG * recipMidgreyB, gammaB) * safeMidgreyB;
            float bAdj_B = std::pow(safeB * recipMidgreyB, gammaB) * safeMidgreyB;
            
            // CYM adjustments - apply each gamma to all RGB channels
            float safeMidgreyC = std::max(0.001f, std::min(0.999f, midgreyC));
            float recipMidgreyC = 1.0f / safeMidgreyC;
            float cAdj_R = std::pow(safeR * recipMidgreyC, gammaC) * safeMidgreyC;
            float cAdj_G = std::pow(safeG * recipMidgreyC, gammaC) * safeMidgreyC;
            float cAdj_B = std::pow(safeB * recipMidgreyC, gammaC) * safeMidgreyC;
            
            float safeMidgreyM = std::max(0.001f, std::min(0.999f, midgreyM));
            float recipMidgreyM = 1.0f / safeMidgreyM;
            float mAdj_R = std::pow(safeR * recipMidgreyM, gammaM) * safeMidgreyM;
            float mAdj_G = std::pow(safeG * recipMidgreyM, gammaM) * safeMidgreyM;
            float mAdj_B = std::pow(safeB * recipMidgreyM, gammaM) * safeMidgreyM;
            
            float safeMidgreyY = std::max(0.001f, std::min(0.999f, midgreyY));
            float recipMidgreyY = 1.0f / safeMidgreyY;
            float yAdj_R = std::pow(safeR * recipMidgreyY, gammaY) * safeMidgreyY;
            float yAdj_G = std::pow(safeG * recipMidgreyY, gammaY) * safeMidgreyY;
            float yAdj_B = std::pow(safeB * recipMidgreyY, gammaY) * safeMidgreyY;
            
            // Mix results - each output channel gets specific adjustment blend
            float finalR = cAdj_R + (rAdj_R - cAdj_R) * tiltCR;
            float finalG = mAdj_G + (gAdj_G - mAdj_G) * tiltGM;
            float finalB = yAdj_B + (bAdj_B - yAdj_B) * tiltBY;

            // Clamp and output
            p_Output[index + 0] = std::max(0.0f, std::min(1.0f, finalR));
            p_Output[index + 1] = std::max(0.0f, std::min(1.0f, finalG));
            p_Output[index + 2] = std::max(0.0f, std::min(1.0f, finalB));
            p_Output[index + 3] = a;
        }
    }
}